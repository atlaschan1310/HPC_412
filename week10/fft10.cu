#include "hip/hip_runtime.h"
#include <blitz/array.h>
#include <complex>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipfft/hipfftw.h>
using namespace blitz;
using std::complex;

void fill_array(Array<float,2> &data) {
    // Set the grid to the sum of two sine functions
    for (int i=0; i < data.rows(); i++) {
        for (int j=0; j < data.cols(); j++) {
            float x = (float)i / 25.0; // Period of 1/4 of the box in x
            float y = (float)j / 10.0; // Period of 1/10 of the box in y
            data(i,j) = sin(2.0 * M_PI * x) + sin(2.0 * M_PI * y);
        }
    }
}

// Verify the FFT (kdata) of data by performing a reverse transform and comparing
bool validate(Array<float,2> &data,Array<std::complex<float>, 2> kdata) {
    Array<float,2> rdata(data.extent());
    fftwf_plan plan = fftwf_plan_dft_c2r_2d(data.rows(), data.cols(),
        reinterpret_cast<fftwf_complex*>(kdata.data()), rdata.data(), FFTW_ESTIMATE);
    fftwf_execute(plan);
    fftwf_destroy_plan(plan);
    rdata /= data.size(); // Normalize for the FFT
    return all(abs(data - rdata) < 1e-5);
}
template<typename T>
bool validate2d(const Array<T,2>& r1, const Array<T,2>& r2) {
	return all(abs(r1-r2) < 1e-5);
}

int main() {
    int n = 10000;
   /*
    // Out of place
    Array<float,2> rdata1(n,n);
    Array<std::complex<float>, 2> kdata1(n, n/2 + 1);
    fftwf_plan plan1  = fftwf_plan_dft_r2c_2d(n, n,
        rdata1.data(), reinterpret_cast<fftwf_complex*>(kdata1.data()), FFTW_ESTIMATE);
    fill_array(rdata1);
    fftwf_execute(plan1);
    fftwf_destroy_plan(plan1);
    std::cout << ">>> Out of place FFT " << (validate(rdata1,kdata1)?"match":"MISMATCH") << endl;

    // in-place
    Array<float,2> raw_data2(n,n+2);
    Array<float,2> rdata2 = raw_data2(Range(0,n-1),Range(0,n-1));
    fftwf_plan plan2  = fftwf_plan_dft_r2c_2d(n, n,
        rdata2.data(), reinterpret_cast<fftwf_complex*>(rdata2.data()), FFTW_ESTIMATE);
    fill_array(rdata2);
    fftwf_execute(plan2);
    fftwf_destroy_plan(plan2);
    Array<std::complex<float>, 2> kdata2(reinterpret_cast<std::complex<float>*>(rdata2.data()),
        shape(n, n/2 + 1),neverDeleteData);
    std::cout << ">>> In-place FFT " << (validate(rdata1,kdata2)?"match":"MISMATCH") << endl;
    //Exercise3   
    Array<float,2> raw_data3(n,n+2);
    Array<float,2> data3 = raw_data3(blitz::Range::all(), blitz::Range(0,n-1));
    fill_array(data3);
    Array<float,2> raw_data4(n,n+2);
    void* device_data;
    size_t size_in_byte = sizeof(float) * n * (n + 2);
    hipMalloc(&device_data, size_in_byte);
    hipMemcpy(device_data, raw_data3.data(), size_in_byte, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipMemcpy(raw_data4.data(), device_data, size_in_byte, hipMemcpyDeviceToHost);
    std::cout << (validate2d(raw_data3, raw_data4) ? "passed" : "failed") << std::endl;
    hipFree(device_data);
    */
    Array<float,2> raw_data5(n,n+2);
    Array<float,2> data5 = raw_data5(blitz::Range::all(), blitz::Range(0,n-1));
    Array<std::complex<float>,2> data6(n, n/2+1);
    float* device_data;
    size_t size_in_byte = sizeof(float) * n * (n + 2);
    hipMalloc(&device_data, size_in_byte);
    hipMemcpy(device_data, raw_data5.data(), size_in_byte, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipfftHandle plan;
    int dim[2] = {n,n};
    int inembed[2] = {n,n+2};
    int onembed[2] = {n, n/2+1};
    int batch = 1;
    int odist = 1;
    int idist = 2;
    int istride = 1;
    int ostride = 1;

    hipfftPlanMany(&plan, 2, dim, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch);
    hipfftExecR2C(plan, device_data, (hipfftComplex*)device_data);
    hipDeviceSynchronize();
    hipfftDestroy(plan);
    hipMemcpy(data6.data(), device_data, size_in_byte, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(device_data);  
    std::cout << ">>> cuda FFT " << (validate(data5,data6)?"match":"MISMATCH") << endl;
    return 0;
 }
