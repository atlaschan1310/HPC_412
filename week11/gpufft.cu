#include "gpufft.h"

static const char *cufftGetErrorString(hipfftResult cufft_error_type)
{

    switch (cufft_error_type)
    {

    case HIPFFT_SUCCESS:
        return "HIPFFT_SUCCESS: The CUFFT operation was performed";

    case HIPFFT_INVALID_PLAN:
        return "HIPFFT_INVALID_PLAN: The CUFFT plan to execute is invalid";

    case HIPFFT_ALLOC_FAILED:
        return "HIPFFT_ALLOC_FAILED: The allocation of data for CUFFT in memory failed";

    case HIPFFT_INVALID_TYPE:
        return "HIPFFT_INVALID_TYPE: The data type used by CUFFT is invalid";

    case HIPFFT_INVALID_VALUE:
        return "HIPFFT_INVALID_VALUE: The data value used by CUFFT is invalid";

    case HIPFFT_INTERNAL_ERROR:
        return "HIPFFT_INTERNAL_ERROR: An internal error occurred in CUFFT";

    case HIPFFT_EXEC_FAILED:
        return "HIPFFT_EXEC_FAILED: The execution of a plan by CUFFT failed";

    case HIPFFT_SETUP_FAILED:
        return "HIPFFT_SETUP_FAILED: The setup of CUFFT failed";

    case HIPFFT_INVALID_SIZE:
        return "HIPFFT_INVALID_SIZE: The size of the data to be used by CUFFT is invalid";

    case HIPFFT_UNALIGNED_DATA:
        return "HIPFFT_UNALIGNED_DATA: The data to be used by CUFFT is unaligned in memory";
    }

    return "Unknown CUFFT Error";
}

void compute_fft_2D_R2C(blitz::Array<float, 2> &grid, void *data)
{
    int n[] = {grid.rows(), grid.cols()}; // 2D FFT of length NxN
    int inembed[] = {grid.rows(), 2 * (grid.cols() / 2 + 1)};
    int onembed[] = {grid.rows(), (grid.cols() / 2 + 1)};
    int howmany = 1;
    int odist = grid.rows() * (grid.cols() / 2 + 1); // Output distance is in "complex"
    int idist = 2 * odist;                           // Input distance is in "real"
    int istride = 1;                                 // Elements of each FFT are adjacent
    int ostride = 1;

    hipfftHandle plan;
/*
    hipfftPlanMany(&plan, sizeof(n) / sizeof(n[0]), n,
                  inembed, istride, idist,
                  onembed, ostride, odist,
                  HIPFFT_R2C, howmany);
*/


    size_t workSize;
    void* workArea;
    hipfftCreate(&plan);
    hipfftSetAutoAllocation(plan,0);
    hipfftMakePlanMany(plan,sizeof(n) / sizeof(n[0]), n,
                  inembed, istride, idist,
                  onembed, ostride, odist,
                  HIPFFT_R2C, howmany, &workSize);
   
    auto data_size = sizeof(hipfftComplex) * howmany * grid.rows() * (grid.cols() / 2 + 1);
    hipMalloc((void**)&workArea, workSize);
    hipfftSetWorkArea(plan, workArea);
    
    hipMemcpy(data, grid.dataFirst(), data_size, hipMemcpyHostToDevice);

    auto status = hipfftExecR2C(plan, reinterpret_cast<hipfftReal *>(data), reinterpret_cast<hipfftComplex *>(data));
    if (status != HIPFFT_SUCCESS)
    {
        const char *errorString = cufftGetErrorString(status);
        printf("CUDA cuFFT Error: %s\n", errorString);
    }

    hipMemcpy(grid.dataFirst(), data, data_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    //    hipFree(data);
    hipfftDestroy(plan);
}

void compute_fft_2D_R2C_Stream(blitz::Array<float, 2> &grid, void *data, hipfftHandle* plan, void* workArea, hipStream_t* stream)
{
    int howmany = 1;
    //    hipfftComplex *data;
    auto data_size = sizeof(hipfftComplex) * howmany * grid.rows() * (grid.cols() / 2 + 1);
    //    hipMalloc((void**)&data, data_size);
    hipMemcpyAsync(data, grid.dataFirst(), data_size, hipMemcpyHostToDevice, *stream);
    hipfftSetStream(*plan, *stream);
    hipfftSetWorkArea(*plan, workArea);
    auto status = hipfftExecR2C(*plan, reinterpret_cast<hipfftReal *>(data), reinterpret_cast<hipfftComplex *>(data));
    if (status != HIPFFT_SUCCESS)
    {
        const char *errorString = cufftGetErrorString(status);
        printf("CUDA cuFFT Error: %s\n", errorString);
    }

    hipMemcpyAsync(grid.dataFirst(), data, data_size, hipMemcpyDeviceToHost, *stream);
    hipDeviceSynchronize();
    //    hipFree(data);
}

void *allocate_cuda_slab(size_t nGrid)
{
    void *cuda_slab;
    auto slab_size = sizeof(hipfftComplex) * nGrid * (nGrid / 2 + 1);
    hipMalloc((void **)&cuda_slab, slab_size);
    return cuda_slab;
}
void* allocate_cuda_slab_Stream(size_t nGrid, hipStream_t* stream) {
    void* cuda_slab;
    auto slab_size = sizeof(hipfftComplex) * nGrid * (nGrid / 2 + 1);
    hipMallocAsync((void**)&cuda_slab, slab_size, *stream);
    return cuda_slab;
}

void* allocate_cuda_size(size_t size) {
    void* cuda_data;
    hipMalloc((void **)&cuda_data, size);
    return cuda_data;
}

void* allocate_cuda_size_Stream(size_t size, hipStream_t* stream) {
    void* cuda_data;
    hipMallocAsync((void **)&cuda_data, size, *stream);
    return cuda_data;
}

void destroy_cuda_data(void* data) {
    hipFree(data);
}

void destroy_cuda_data_Stream(void* data, hipStream_t* stream) {
    hipFreeAsync(data, *stream);
}
